#include "hip/hip_runtime.h"
/*
 * This kernel uses a global memory approach, where each thread is responsible for computing a single element in the resulting vector. 
 *
 */
#include "axpy.h"

__global__ void axpy_cudakernel_P3(int N, REAL *Y, const REAL *X, REAL a) {
    __shared__ REAL shared_X[256];
    __shared__ REAL shared_Y[256];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        shared_X[threadIdx.x] = X[i];  // Load data into shared memory
        shared_Y[threadIdx.x] = Y[i];
        __syncthreads();  // Synchronize threads in a block

        #pragma unroll
        for (int j = 0; j < blockDim.x; j++) {
            shared_Y[j] += a * shared_X[j];  // Perform AXPY in shared memory
        }

        Y[i] = shared_Y[threadIdx.x];  // Store the result back to global memory
    }
}

void axpy_kernel(int N, REAL* Y, REAL* X, REAL a) {
    REAL *d_x, *d_y;
    hipMalloc(&d_x, N*sizeof(REAL));
    hipMalloc(&d_y, N*sizeof(REAL));

    hipMemcpy(d_x, X, N*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_y, Y, N*sizeof(REAL), hipMemcpyHostToDevice);

    axpy_cudakernel_P3<<<(N+255)/256, 256>>>(d_x, d_y, N, a);
    hipDeviceSynchronize();

    hipMemcpy(Y, d_y, N*sizeof(REAL), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
}

