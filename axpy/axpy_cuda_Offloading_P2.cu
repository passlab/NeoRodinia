#include "hip/hip_runtime.h"
/*
 * This kernel uses a global memory approach, where each thread is responsible for computing a single element in the resulting vector. 
 *
 */
#include "axpy.h"

__global__ void axpy_cudakernel_P2(int N, REAL *Y, const REAL *X, REAL a) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        Y[i] += a * X[i];  // Ensures coalesced access
    }
}

void axpy_kernel(int N, REAL* Y, REAL* X, REAL a) {
    REAL *d_x, *d_y;
    hipMalloc(&d_x, N*sizeof(REAL));
    hipMalloc(&d_y, N*sizeof(REAL));

    hipMemcpy(d_x, X, N*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_y, Y, N*sizeof(REAL), hipMemcpyHostToDevice);

    axpy_cudakernel_P2<<<(N+255)/256, 256>>>(d_x, d_y, N, a);
    hipDeviceSynchronize();

    hipMemcpy(Y, d_y, N*sizeof(REAL), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
}

