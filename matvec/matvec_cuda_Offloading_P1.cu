#include "hip/hip_runtime.h"
/*
 * This kernel provides a straightforward mapping of threads to rows, with each thread handling a separate row. 
 * This approach may be more suitable when the number of rows is small or when the matrix is wider than it is tall (m is larger than n).
 *
 */
#include "matvec.h"

__global__ void matvec_P1(REAL *matrix, REAL *vector, REAL *result, int n, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        REAL temp = 0.0;
        for (int j = 0; j < m; j++)
            temp += matrix[i * m + j] * vector[j];
        result[i] = temp;
    }
}

void matvec_kernel(int N, REAL *A, REAL *B, REAL *C) {
    REAL *d_matrix, *d_vector, *d_result;
    hipMalloc(&d_matrix, N * N * sizeof(REAL));
    hipMalloc(&d_vector, N * sizeof(REAL));
    hipMalloc(&d_result, N * sizeof(REAL));

    hipMemcpy(d_matrix, A, N * N * sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, B, N * sizeof(REAL), hipMemcpyHostToDevice);

    // Perform matvec elements
    int blockSize = 1024;
    int gridSize = (N + blockSize - 1) / blockSize;
    matvec_P1<<<gridSize, blockSize>>>(d_matrix, d_vector, d_result, N, N);

    hipMemcpy(C, d_result, N * sizeof(REAL), hipMemcpyDeviceToHost);
    hipFree(d_matrix);
    hipFree(d_vector);
    hipFree(d_result);
}
