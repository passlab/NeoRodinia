#include "hip/hip_runtime.h"
/*
 * This kernel uses a 2D grid of blocks and 2D blocks of threads. 
 *
 */
#include "matmul.h"
#define BLOCK_SIZE 16

__global__ void global_block(REAL* A, REAL* B, REAL* C, int n) {
    int wA = n;
    int wB = n;

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += A[a + wA * ty + k] * B[b + wB * k + tx];
        }
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

void matmul_kernel(int N, REAL* A, REAL* B, REAL* C) {
    REAL *A_device, *B_device, *C_device;
    hipMalloc(&A_device, N*N*sizeof(REAL));
    hipMalloc(&B_device, N*N*sizeof(REAL));
    hipMalloc(&C_device, N*N*sizeof(REAL));

    hipMemcpy(A_device, A, N*N*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B, N*N*sizeof(REAL), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);
    global_block<<<dimGrid, dimBlock>>>(A_device, B_device, C_device, N);

    hipMemcpy(C, C_device, N*N*sizeof(REAL), hipMemcpyDeviceToHost);
    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);
}