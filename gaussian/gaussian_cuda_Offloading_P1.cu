#include "hip/hip_runtime.h"
/*
 * Serial Version
 *
 */
#include "gaussian.h"
#include "hip/hip_runtime.h"

void checkCUDAError(const char *msg);

/*-------------------------------------------------------
 ** Fan1() -- Calculate multiplier matrix
 ** Pay attention to the index. Index i gives the range
 ** which starts from 0 to range-1. The real values of
 ** the index should be adjusted and related to the value
 ** of t which is defined in the ForwardSub().
 **-------------------------------------------------------
 */
__global__ void Fan1_cuda(float *m_cuda, float *a_cuda, int Size, int t)
{   
	//if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) printf(".");
	//printf("blockIDx.x:%d,threadIdx.x:%d,Size:%d,t:%d,Size-1-t:%d\n",blockIdx.x,threadIdx.x,Size,t,Size-1-t);

	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	*(m_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) = *(a_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) / *(a_cuda+Size*t+t);
}

/*-------------------------------------------------------
 ** Fan2() -- Modify the matrix A into LUD
 **-------------------------------------------------------
 */
__global__ void Fan2_cuda(float *m_cuda, float *a_cuda, float *b_cuda,int Size, int j1, int t)
{
	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	if(threadIdx.y + blockIdx.y * blockDim.y >= Size-t) return;

	int xidx = blockIdx.x * blockDim.x + threadIdx.x;
	int yidx = blockIdx.y * blockDim.y + threadIdx.y;
	//printf("blockIdx.x:%d,threadIdx.x:%d,blockIdx.y:%d,threadIdx.y:%d,blockDim.x:%d,blockDim.y:%d\n",blockIdx.x,threadIdx.x,blockIdx.y,threadIdx.y,blockDim.x,blockDim.y);

	a_cuda[Size*(xidx+1+t)+(yidx+t)] -= m_cuda[Size*(xidx+1+t)+t] * a_cuda[Size*t+(yidx+t)];
	//a_cuda[xidx+1+t][yidx+t] -= m_cuda[xidx+1+t][t] * a_cuda[t][yidx+t];
	if(yidx == 0){
		//printf("blockIdx.x:%d,threadIdx.x:%d,blockIdx.y:%d,threadIdx.y:%d,blockDim.x:%d,blockDim.y:%d\n",blockIdx.x,threadIdx.x,blockIdx.y,threadIdx.y,blockDim.x,blockDim.y);
		//printf("xidx:%d,yidx:%d\n",xidx,yidx);
		b_cuda[xidx+1+t] -= m_cuda[Size*(xidx+1+t)+(yidx+t)] * b_cuda[t];
	}
}

/*------------------------------------------------------
 ** ForwardSub() -- Forward substitution of Gaussian
 ** elimination.
 **------------------------------------------------------
 */
void ForwardSub(int Size, float *a, float *b, float *m) {
	int t;
    float *m_cuda,*a_cuda,*b_cuda;

	// allocate memory on GPU
	hipMalloc((void **) &m_cuda, Size * Size * sizeof(float));

	hipMalloc((void **) &a_cuda, Size * Size * sizeof(float));

	hipMalloc((void **) &b_cuda, Size * sizeof(float));	

	// copy memory to GPU
	hipMemcpy(m_cuda, m, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(a_cuda, a, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(b_cuda, b, Size * sizeof(float),hipMemcpyHostToDevice );

	int block_size,grid_size;

	block_size = MAXBLOCKSIZE;
	grid_size = (Size/block_size) + (!(Size%block_size)? 0:1);
	//printf("1d grid size: %d\n",grid_size);


	dim3 dimBlock(block_size);
	dim3 dimGrid(grid_size);
	//dim3 dimGrid( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	int blockSize2d, gridSize2d;
	blockSize2d = BLOCK_SIZE_XY;
	gridSize2d = (Size/blockSize2d) + (!(Size%blockSize2d?0:1)); 

	dim3 dimBlockXY(blockSize2d,blockSize2d);
	dim3 dimGridXY(gridSize2d,gridSize2d);

	for (t=0; t<(Size-1); t++) {
		Fan1_cuda<<<dimGrid,dimBlock>>>(m_cuda,a_cuda,Size,t);
		hipDeviceSynchronize();
		Fan2_cuda<<<dimGridXY,dimBlockXY>>>(m_cuda,a_cuda,b_cuda,Size,Size-t,t);
		hipDeviceSynchronize();
		checkCUDAError("Fan2");
	}

	// copy memory back to CPU
	hipMemcpy(m, m_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(a, a_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(b, b_cuda, Size * sizeof(float),hipMemcpyDeviceToHost );
	hipFree(m_cuda);
	hipFree(a_cuda);
	hipFree(b_cuda);
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}