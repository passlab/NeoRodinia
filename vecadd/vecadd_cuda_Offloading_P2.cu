#include "hip/hip_runtime.h"
/*
 * This kernel uses a global memory approach, where each thread is responsible for computing a single element in the resulting vector. 
 *
 */
#include "vecadd.h"

__global__ void vecadd_cudakernel_P2(int N, REAL *Y, const REAL *X) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        Y[i] += X[i];  // Ensures coalesced access
    }
}

void vecadd_kernel(int N, REAL* Y, REAL* X) {
    REAL *d_x, *d_y;
    hipMalloc(&d_x, N*sizeof(REAL));
    hipMalloc(&d_y, N*sizeof(REAL));

    hipMemcpy(d_x, X, N*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_y, Y, N*sizeof(REAL), hipMemcpyHostToDevice);

    vecadd_cudakernel_P2<<<(N+255)/256, 256>>>(N, d_x, d_y);
    hipDeviceSynchronize();

    hipMemcpy(Y, d_y, N*sizeof(REAL), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
}

